#include "hip/hip_runtime.h"
#include "v1/kernels.cuh"

__global__ void coalesced_fp32gemm(
    float* A, float* B, float* C, 
    int M, int N, int K, 
    float alpha, float beta
) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M and col < N)
    {
        float s = 0.0;
        for (int k = 0; k < K; k++)
        {
            s += A[row * K + k] * B[col + k * N];
        }
        C[row * N + col] = alpha * s + beta * C[row * N + col];
    }
}