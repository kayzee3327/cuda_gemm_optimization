#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void coalesced_fp32gemm(float* A, float* B, float* C, 
    int M, int N, int K, float alpha, float beta) 
{
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M and col < N)
    {
        float s = 0.0;
        for (size_t k = 0; k < K; k++)
        {
            s += A[row * K + k] * B[col + k * N];
        }
        C[row * N + col] = alpha * s + beta * C[row * N + col];
    }
}